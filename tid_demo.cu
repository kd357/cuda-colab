#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void myKernel(int *d_a)
{
    int tx = threadIdx.x;
    d_a[tx] += 1;
    printf("Hello, world from the device!\n");
}
int main()
{
    int *a = (int *)malloc(sizeof(int) * 10);
    int *d_a;
    int i;
    for (i = 0; i < 10; i++)
        a[i] = i;
    for (i = 0; i < 10; i++)
        printf("%d,", a[i]);
    printf("\n");
    hipMalloc((void **)&d_a, 10 * sizeof(int));
    hipMemcpy(d_a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
    myKernel<<<1, 10>>>(d_a);
    hipMemcpy(a, d_a, 10 * sizeof(int), hipMemcpyDeviceToHost);
    for (i = 0; i < 10; i++)
        printf("%d,", a[i]);
    printf("\n");

    hipFree(d_a);
}
